#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>

// CUDA kernel for calculating Levenshtein distances in parallel
__global__ void levenshtein_kernel(const char* d_strings, const size_t* d_lengths, 
                                 const size_t* d_offsets, const char* d_query, 
                                 size_t query_length, size_t* d_results, size_t strings_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= strings_count) return;
    
    // Get string from array using offset
    const char* str = &d_strings[d_offsets[idx]];
    size_t str_length = d_lengths[idx];
    
    // Handle base cases
    if (query_length == 0) {
        d_results[idx] = str_length;
        return;
    }
    if (str_length == 0) {
        d_results[idx] = query_length;
        return;
    }
    
    // Allocate cache in local memory
    size_t* cache;
    cache = new size_t[str_length];
    
    // Initialize cache
    for (size_t i = 0; i < str_length; i++) {
        cache[i] = i + 1;
    }
    
    // Compute Levenshtein distance
    size_t distance, bDistance, result;
    for (size_t bIndex = 0; bIndex < query_length; bIndex++) {
        char code = d_query[bIndex];
        result = distance = bIndex;
        
        for (size_t index = 0; index < str_length; index++) {
            bDistance = code == str[index] ? distance : distance + 1;
            distance = cache[index];
            
            cache[index] = result = distance > result
                ? bDistance > result ? result + 1 : bDistance
                : bDistance > distance ? distance + 1 : bDistance;
        }
    }
    
    d_results[idx] = result;
    delete[] cache;
}

// CUDA kernel for filtering strings based on search criteria
__global__ void filter_kernel(const char* d_strings, const char* d_strings_sans_accent, 
                            const char* d_strings_majuscule, const size_t* d_lengths, 
                            const size_t* d_offsets, const char* d_query, size_t query_length, 
                            int search_type, int* d_results, size_t strings_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= strings_count) return;
    
    // Get strings using offsets
    const char* str = &d_strings[d_offsets[idx]];
    const char* str_sans_accent = &d_strings_sans_accent[d_offsets[idx]];
    const char* str_majuscule = &d_strings_majuscule[d_offsets[idx]];
    size_t str_length = d_lengths[idx];
    
    bool match = false;
    
    if (search_type == 0) { // "Commencant par"
        if (str_length >= query_length) {
            bool match_standard = true;
            bool match_sans_accent = true;
            bool match_majuscule = true;
            
            for (size_t i = 0; i < query_length; i++) {
                if (str[i] != d_query[i]) match_standard = false;
                if (str_sans_accent[i] != d_query[i]) match_sans_accent = false;
                if (str_majuscule[i] != d_query[i]) match_majuscule = false;
            }
            
            match = match_standard || match_sans_accent || match_majuscule;
        }
    } 
    else if (search_type == 1) { // "Finissant par"
        if (str_length >= query_length) {
            size_t offset = str_length - query_length;
            bool match_standard = true;
            bool match_sans_accent = true;
            bool match_majuscule = true;
            
            for (size_t i = 0; i < query_length; i++) {
                if (str[offset + i] != d_query[i]) match_standard = false;
                if (str_sans_accent[offset + i] != d_query[i]) match_sans_accent = false;
                if (str_majuscule[offset + i] != d_query[i]) match_majuscule = false;
            }
            
            match = match_standard || match_sans_accent || match_majuscule;
        }
    }
    else { // "Contenant"
        for (size_t i = 0; i <= str_length - query_length; i++) {
            bool match_standard = true;
            bool match_sans_accent = true;
            bool match_majuscule = true;
            
            for (size_t j = 0; j < query_length; j++) {
                if (str[i + j] != d_query[j]) match_standard = false;
                if (str_sans_accent[i + j] != d_query[j]) match_sans_accent = false;
                if (str_majuscule[i + j] != d_query[j]) match_majuscule = false;
                
                if (!match_standard && !match_sans_accent && !match_majuscule) break;
            }
            
            if (match_standard || match_sans_accent || match_majuscule) {
                match = true;
                break;
            }
        }
    }
    
    d_results[idx] = match ? 1 : 0;
}

// Host helper functions to manage device memory and call kernels

// GPU implementation of levenshtein_n
extern "C" size_t cuda_levenshtein_n(const char *a, const size_t length, const char *b, const size_t bLength) {
    // Handle base cases
    if (a == b) return 0;
    if (!length) return bLength;
    if (!bLength) return length;
    
    // Allocate device memory
    char *d_a, *d_b;
    size_t *d_result, *d_length, *d_offset;
    hipMalloc(&d_a, length);
    hipMalloc(&d_b, bLength);
    hipMalloc(&d_result, sizeof(size_t));
    hipMalloc(&d_length, sizeof(size_t));
    hipMalloc(&d_offset, sizeof(size_t));
    
    // Copy data to device
    hipMemcpy(d_a, a, length, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bLength, hipMemcpyHostToDevice);
    
    size_t zero = 0;
    hipMemcpy(d_offset, &zero, sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(d_length, &length, sizeof(size_t), hipMemcpyHostToDevice);
    
    // Launch kernel with a single thread
    levenshtein_kernel<<<1, 1>>>(d_a, d_length, d_offset, d_b, bLength, d_result, 1);
    
    // Get result
    size_t h_result;
    hipMemcpy(&h_result, d_result, sizeof(size_t), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
    hipFree(d_length);
    hipFree(d_offset);
    
    return h_result;
}

// GPU implementation of levenshtein
extern "C" size_t cuda_levenshtein(const char *a, const char *b) {
    return cuda_levenshtein_n(a, strlen(a), b, strlen(b));
}

// GPU implementation of calculate_distances
extern "C" void cuda_calculate_distances(const char **names, size_t names_count, const char *query, size_t *distances, size_t min_distance, size_t max_suggestions) {
    // Prepare data for GPU
    size_t query_len = strlen(query);
    
    // Calculate total size needed for string data
    size_t total_size = 0;
    size_t* lengths = (size_t*)malloc(names_count * sizeof(size_t));
    size_t* offsets = (size_t*)malloc(names_count * sizeof(size_t));
    
    for (size_t i = 0; i < names_count; i++) {
        lengths[i] = strlen(names[i]);
        offsets[i] = total_size;
        total_size += lengths[i];
    }
    
    // Allocate memory for flattened string array
    char* h_flat_strings = (char*)malloc(total_size);
    
    // Flatten string array
    for (size_t i = 0; i < names_count; i++) {
        memcpy(h_flat_strings + offsets[i], names[i], lengths[i]);
    }
    
    // Allocate device memory
    char *d_flat_strings, *d_query;
    size_t *d_lengths, *d_offsets, *d_results, *d_temp_distances;
    
    hipMalloc(&d_flat_strings, total_size);
    hipMalloc(&d_query, query_len);
    hipMalloc(&d_lengths, names_count * sizeof(size_t));
    hipMalloc(&d_offsets, names_count * sizeof(size_t));
    hipMalloc(&d_results, names_count * sizeof(size_t));
    hipMalloc(&d_temp_distances, names_count * sizeof(size_t));
    
    // Copy data to device
    hipMemcpy(d_flat_strings, h_flat_strings, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, query_len, hipMemcpyHostToDevice);
    hipMemcpy(d_lengths, lengths, names_count * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets, names_count * sizeof(size_t), hipMemcpyHostToDevice);
    
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (names_count + threadsPerBlock - 1) / threadsPerBlock;
    
    levenshtein_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_flat_strings, d_lengths, d_offsets, d_query, query_len, d_temp_distances, names_count
    );
    
    // Copy results back
    size_t* temp_distances = (size_t*)malloc(names_count * sizeof(size_t));
    hipMemcpy(temp_distances, d_temp_distances, names_count * sizeof(size_t), hipMemcpyDeviceToHost);
    
    // Process results on CPU side
    size_t suggestions_count = 0;
    for (size_t i = 0; i < names_count; i++) {
        if (temp_distances[i] < min_distance && suggestions_count < max_suggestions) {
            distances[suggestions_count++] = temp_distances[i];
        }
    }
    
    // Clean up
    free(h_flat_strings);
    free(lengths);
    free(offsets);
    free(temp_distances);
    
    hipFree(d_flat_strings);
    hipFree(d_query);
    hipFree(d_lengths);
    hipFree(d_offsets);
    hipFree(d_results);
    hipFree(d_temp_distances);
}

// GPU implementation of calculate_final_distances
extern "C" void cuda_calculate_final_distances(const char **names, size_t names_count, const char *query, size_t *distances) {
    // Prepare data similar to cuda_calculate_distances
    size_t query_len = strlen(query);
    
    // Calculate total size needed for string data
    size_t total_size = 0;
    size_t* lengths = (size_t*)malloc(names_count * sizeof(size_t));
    size_t* offsets = (size_t*)malloc(names_count * sizeof(size_t));
    
    for (size_t i = 0; i < names_count; i++) {
        lengths[i] = strlen(names[i]);
        offsets[i] = total_size;
        total_size += lengths[i];
    }
    
    // Allocate memory for flattened string array
    char* h_flat_strings = (char*)malloc(total_size);
    
    // Flatten string array
    for (size_t i = 0; i < names_count; i++) {
        memcpy(h_flat_strings + offsets[i], names[i], lengths[i]);
    }
    
    // Allocate device memory
    char *d_flat_strings, *d_query;
    size_t *d_lengths, *d_offsets, *d_distances;
    
    hipMalloc(&d_flat_strings, total_size);
    hipMalloc(&d_query, query_len);
    hipMalloc(&d_lengths, names_count * sizeof(size_t));
    hipMalloc(&d_offsets, names_count * sizeof(size_t));
    hipMalloc(&d_distances, names_count * sizeof(size_t));
    
    // Copy data to device
    hipMemcpy(d_flat_strings, h_flat_strings, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, query_len, hipMemcpyHostToDevice);
    hipMemcpy(d_lengths, lengths, names_count * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets, names_count * sizeof(size_t), hipMemcpyHostToDevice);
    
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (names_count + threadsPerBlock - 1) / threadsPerBlock;
    
    levenshtein_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_flat_strings, d_lengths, d_offsets, d_query, query_len, d_distances, names_count
    );
    
    // Copy results back
    hipMemcpy(distances, d_distances, names_count * sizeof(size_t), hipMemcpyDeviceToHost);
    
    // Clean up
    free(h_flat_strings);
    free(lengths);
    free(offsets);
    
    hipFree(d_flat_strings);
    hipFree(d_query);
    hipFree(d_lengths);
    hipFree(d_offsets);
    hipFree(d_distances);
}

// GPU implementation of filter_df
extern "C" void cuda_filter_df(const char **names, const char **names_sans_accent, const char **names_majuscule, 
                            size_t names_count, const char *query, const char *search_type, int *results) {
    // Determine search type code
    int search_type_code;
    if (strcmp(search_type, "Commencant par") == 0)
        search_type_code = 0;
    else if (strcmp(search_type, "Finissant par") == 0)
        search_type_code = 1;
    else
        search_type_code = 2; // "Contenant"
    
    size_t query_len = strlen(query);
    
    // Calculate total size needed for string data
    size_t total_size = 0;
    size_t* lengths = (size_t*)malloc(names_count * sizeof(size_t));
    size_t* offsets = (size_t*)malloc(names_count * sizeof(size_t));
    
    for (size_t i = 0; i < names_count; i++) {
        lengths[i] = strlen(names[i]);
        offsets[i] = total_size;
        total_size += lengths[i];
    }
    
    // Allocate memory for flattened string arrays
    char* h_flat_strings = (char*)malloc(total_size);
    char* h_flat_sans_accent = (char*)malloc(total_size);
    char* h_flat_majuscule = (char*)malloc(total_size);
    
    // Flatten string arrays
    for (size_t i = 0; i < names_count; i++) {
        memcpy(h_flat_strings + offsets[i], names[i], lengths[i]);
        memcpy(h_flat_sans_accent + offsets[i], names_sans_accent[i], lengths[i]);
        memcpy(h_flat_majuscule + offsets[i], names_majuscule[i], lengths[i]);
    }
    
    // Allocate device memory
    char *d_flat_strings, *d_flat_sans_accent, *d_flat_majuscule, *d_query;
    size_t *d_lengths, *d_offsets;
    int *d_results;
    
    hipMalloc(&d_flat_strings, total_size);
    hipMalloc(&d_flat_sans_accent, total_size);
    hipMalloc(&d_flat_majuscule, total_size);
    hipMalloc(&d_query, query_len);
    hipMalloc(&d_lengths, names_count * sizeof(size_t));
    hipMalloc(&d_offsets, names_count * sizeof(size_t));
    hipMalloc(&d_results, names_count * sizeof(int));
    
    // Copy data to device
    hipMemcpy(d_flat_strings, h_flat_strings, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_flat_sans_accent, h_flat_sans_accent, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_flat_majuscule, h_flat_majuscule, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_query, query, query_len, hipMemcpyHostToDevice);
    hipMemcpy(d_lengths, lengths, names_count * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets, names_count * sizeof(size_t), hipMemcpyHostToDevice);
    
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (names_count + threadsPerBlock - 1) / threadsPerBlock;
    
    filter_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_flat_strings, d_flat_sans_accent, d_flat_majuscule, 
        d_lengths, d_offsets, d_query, query_len, 
        search_type_code, d_results, names_count
    );
    
    // Copy results back
    hipMemcpy(results, d_results, names_count * sizeof(int), hipMemcpyDeviceToHost);
    
    // Clean up
    free(h_flat_strings);
    free(h_flat_sans_accent);
    free(h_flat_majuscule);
    free(lengths);
    free(offsets);
    
    hipFree(d_flat_strings);
    hipFree(d_flat_sans_accent);
    hipFree(d_flat_majuscule);
    hipFree(d_query);
    hipFree(d_lengths);
    hipFree(d_offsets);
    hipFree(d_results);
}
